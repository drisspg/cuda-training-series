
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){

  dim3 grid(2);
  dim3 block(2);
  hello<<<grid, block>>>();
  hipDeviceSynchronize();
}

